
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>

__global__ // the function is a kernel now
void add(int n, float* x, float* y)
{
	int index = threadIdx.x;
	int stride = blockDim.x;
	for (int i = index; i < n; i += stride) {
		y[i] = x[i] + y[i];
	}
}

int main()
{
	int N = 1 << 20;
	float* x, * y;

	// allocate unified memory -- accesible from CPU or GPU
	hipMallocManaged(&x, N * sizeof(float));
	hipMallocManaged(&y, N * sizeof(float));

	// init the arrays on the host
	for (int i = 0; i < N; i++)
	{
		x[i] = 1.0f;
		y[i] = 2.0f;
	}

	// run kernel on 1M elements on the GPU
	add <<<1, 256>>> (N, x, y);
	
	// wait for gpu to finish
	hipDeviceSynchronize();

	float maxError = 0.0f;
	for (int i = 0; i < N; i++)
	{
		maxError = fmax(maxError, fabs(y[i] - 3.0f));
	}
	std::cout << "Max error: " << maxError << std::endl;

	// free memory
	hipFree(x);
	hipFree(y);

	return 0;
}
