#include "hip/hip_runtime.h"
#include "fully_connected_layer.h"
#include <random>
#include <hip/hip_runtime.h>
#include <>

__global__ void fullyConnectedLayerForward(float* W, float* A, float* Z, float* b, int W_x_dim, int W_y_dim, int A_x_dim, int A_y_dim)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	int Z_x_dim = A_x_dim;
	int Z_y_dim = W_y_dim;

	float Z_value = 0;

	if (row < Z_y_dim && col < Z_x_dim)
	{
		for (int i = 0; i < W_x_dim; i++)
		{
			Z_value += W[row * W_x_dim + i] * A[i * A_x_dim + col];
		}
		Z[row * Z_x_dim + col] = Z_value + b[row];
	}
}

__global__ void fullyConnectedLayerBackprop(float* W, float* dZ, float* dA, int W_x_dim, int W_y_dim, int dZ_x_dim, int dZ_y_dim)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	int dA_x_dim = dZ_x_dim;
	int dA_y_dim = W_x_dim;

	float dA_value = 0.0f;
	
	if (row < dA_y_dim && col < dA_x_dim)
	{
		for (int i = 0; i < W_y_dim; i++)
		{
			dA_value += W[i * W_x_dim + row] * dZ[i * dZ_x_dim + col];
		}
		dA[row * dA_x_dim + col] = dA_value;
	}
}

void FullyConnectedLayer::initializeBiasWithZeros()
{
	for (int x = 0; x < b.shape.x; x++)
	{
		b[x] = 0;
	}

	b.copyHostToDevice();
}

void FullyConnectedLayer::initializeWeightsRandomly()
{
	std::default_random_engine generator;
	std::normal_distribution<float> normal_distribution(0.0, 1.0);

	for (int x = 0; x < W.shape.x; x++)
	{
		for (int y = 0; y < W.shape.y; y++)
		{
			W[y * W.shape.x + x] = normal_distribution(generator) * weights_init_threshold;
		}
	}

	W.copyHostToDevice();
}

FullyConnectedLayer::FullyConnectedLayer(std::string name, Shape W_shape)
	:W(W_shape), b(W_shape.y, 1)
{
	this->name = name;
	b.allocateMemory();
	W.allocateMemory();
	initializeBiasWithZeros();
	initializeWeightsRandomly();
}

FullyConnectedLayer::~FullyConnectedLayer()
{
}

int FullyConnectedLayer::getXDim() const
{
	return W.shape.x;
}

int FullyConnectedLayer::getYDim() const
{
	return W.shape.y;
}

Tensor2D FullyConnectedLayer::getWeightsMatrix() const
{
	return W;
}

Tensor2D FullyConnectedLayer::getBiasVector() const
{
	return b;
}
